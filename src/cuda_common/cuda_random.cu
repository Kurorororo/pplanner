#include "hip/hip_runtime.h"
#include "random_walk/gpu_rand.h"

namespace pplanner {

__global__
void SetupStates(unsigned long long seed, hiprandState_t *d_states) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &d_states[id]);
}

} // namespace pplanner
